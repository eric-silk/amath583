#include "hip/hip_runtime.h"
#include "Timer.hpp"
#include <assert.h>
#include <iostream>
#include <vector>

#undef NDEBUG

// Sequential norm, for validation
float norm(const std::vector<float>& v) {
  float sum = 0.0;
  for (size_t i = 0; i < v.size(); ++i){
    sum += v[i] * v[i];
  }
  return std::sqrt(sum);
}

template <class T>
__global__ void
noblock_sq(unsigned int array_size, T *x, T *y){
  size_t i = blockDim.x * blockIdx.x + threadIdx.x;
  if (i<array_size) y[i] = x[i] * x[i];
}

__global__ void
sq(float *g_idata, float *g_odata){
  extern __shared__ float sdata[];

  size_t tid = threadIdx.x;
  size_t i = blockIdx.x * blockDim.x + threadIdx.x;

  sdata[tid] = (g_idata[i] * g_idata[i]);
  __syncthreads();

  for (size_t s = 1; s<blockDim.x; s*=2){
    size_t index = 2*s*tid;
    if (index < blockDim.x) {
      sdata[index] += sdata[index + s];
    }
    __syncthreads();
  }

  if (tid == 0){
    g_odata[blockIdx.x] = sdata[0];
  }
}


int main(int argc, char* argv[]) {

  size_t exponent           = 27;
  size_t num_trips          = 1;

  if (argc >= 2) exponent   = std::stol(argv[1]);
  if (argc >= 3) num_trips  = std::stol(argv[2]);

  const size_t N = 1 << exponent;
  int block_size = 256;
  int num_blocks = (N + block_size - 1) / block_size;

  float *x = nullptr, *y = nullptr;

  hipMallocManaged(&x, N*sizeof(float));
  hipMallocManaged(&y, num_blocks*sizeof(float));

  for (int i = 0; i < N; i++) {
    x[i] = i;
  }

  std::vector<float> sequential_vector(x, x+N);
  for (int i = 0; i < N; i++)
  {
    assert(x[i] == sequential_vector[i]);
  }

  DEF_TIMER(cuda_norm);
  START_TIMER(cuda_norm);
  hipDeviceSynchronize();
  for (size_t i = 0; i < num_trips; ++i) {
    sq<<<num_blocks, block_size, block_size*sizeof(float)>>>(x, y);
    hipDeviceSynchronize();
  }
  /* write me: final step, copy out values from y and add on cpu */
  float result = 0.0;
  for (size_t i = 0; i < num_blocks; ++i)
  {
    result += y[i];
  }

  result = std::sqrt(result);

  double cuda_time = STOP_TIMER_QUIETLY(cuda_norm);
  std::cout << exponent << "\t" << num_trips << "\t" << cuda_time << "\t" << result << std::endl;

  // this WILL fail for exponents above 8 or so. Floating point error is a bitch.
  assert(norm(sequential_vector) == result);

  hipFree(x); hipFree(y);
  
  return 0;
}

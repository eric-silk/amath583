#include "hip/hip_runtime.h"
#include "Timer.hpp"
#include <iostream>


__global__
void sq(int n, float *x, float *y){
  /* Write me -- kernel */
  for (size_t i = 0; i < n; ++i)
  {
    y[i] = x[i] * x[i];
  }
}


int main(int argc, char* argv[]) {

  size_t exponent           = 27;
  size_t num_trips          = 1;

  if (argc >= 2) exponent   = std::stol(argv[1]);
  if (argc >= 3) num_trips  = std::stol(argv[2]);

  size_t N = 1 << exponent;
  int block_size = 256;
  int num_blocks = (N + block_size - 1) / block_size;

  float *x = nullptr, *y = nullptr;

  hipMallocManaged(&x, N*sizeof(float));
  hipMallocManaged(&y, num_blocks*sizeof(float));

  for (int i = 0; i < N; i++) {
    x[i] = i;
  }

  DEF_TIMER(cuda_norm);
  START_TIMER(cuda_norm);
  hipDeviceSynchronize();
  for (size_t i = 0; i < num_trips; ++i) {
    /* write me -- launch sq kernel */
    sq<<<num_blocks, block_size>>>(N, x, y);
    hipDeviceSynchronize();
  }
  /* write me: final step, copy out values from y and add on cpu */
  float result = 0.0;
  for (size_t i = 0; i < N; ++i)
  {
    result += y[i];
  }

  result = std::sqrt(result);

  double cuda_time = STOP_TIMER_QUIETLY(cuda_norm);
  std::cout << exponent << "\t" << num_trips << "\t" << cuda_time << std::endl;

  hipFree(x); hipFree(y);
  
  return 0;
}

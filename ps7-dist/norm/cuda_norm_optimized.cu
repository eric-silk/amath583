#include "hip/hip_runtime.h"
#include "Timer.hpp"
#include <iostream>

//..."borrowed" from the reduction examples.
#include "../include/reduction.h"
#include <hip/hip_cooperative_groups.h>

namespace cg = cooperative_groups;

// Utility class used to avoid linker errors with extern
// unsized shared memory arrays with templated type
template<class T>
struct SharedMemory
{
    __device__ inline operator       T *()
    {
        extern __shared__ int __smem[];
        return (T *)__smem;
    }

    __device__ inline operator const T *() const
    {
        extern __shared__ int __smem[];
        return (T *)__smem;
    }
};

/*
    Parallel sum reduction using shared memory
    - takes log(n) steps for n input elements
    - uses n threads
    - only works for power-of-2 arrays
*/

/* This reduction interleaves which threads are active by using the modulo
   operator.  This operator is very expensive on GPUs, and the interleaved
   inactivity means that no whole warps are active, which is also very
   inefficient */
template <class T>
__global__ void
reduce0(T *g_idata, T *g_odata, unsigned int n)
{
    // Handle to thread block group
    cg::thread_block cta = cg::this_thread_block();
    T *sdata = SharedMemory<T>();

    // load shared mem
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;

    sdata[tid] = (i < n) ? g_idata[i] : 0;

    cg::sync(cta);

    // do reduction in shared mem
    for (unsigned int s=1; s < blockDim.x; s *= 2)
    {
        // modulo arithmetic is slow!
        if ((tid % (2*s)) == 0)
        {
            sdata[tid] += sdata[tid + s];
        }

        cg::sync(cta);
    }

    // write result for this block to global mem
    if (tid == 0) g_odata[blockIdx.x] = sdata[0];
}

/*
    This version adds multiple elements per thread sequentially.  This reduces the overall
    cost of the algorithm while keeping the work complexity O(n) and the step complexity O(log n).
    (Brent's Theorem optimization)

    Note, this kernel needs a minimum of 64*sizeof(T) bytes of shared memory.
    In other words if blockSize <= 32, allocate 64*sizeof(T) bytes.
    If blockSize > 32, allocate blockSize*sizeof(T) bytes.
*/
template <class T, unsigned int blockSize, bool nIsPow2>
__global__ void
reduce6(T *g_idata, T *g_odata, size_t n)
{
    // Handle to thread block group
    cg::thread_block cta = cg::this_thread_block();
    T *sdata = SharedMemory<T>();

    // perform first level of reduction,
    // reading from global memory, writing to shared memory
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x*blockSize*2 + threadIdx.x;
    unsigned int gridSize = blockSize*2*gridDim.x;

    T mySum = 0;

    // we reduce multiple elements per thread.  The number is determined by the
    // number of active thread blocks (via gridDim).  More blocks will result
    // in a larger gridSize and therefore fewer elements per thread
    while (i < n)
    {
        mySum += g_idata[i];

        // ensure we don't read out of bounds -- this is optimized away for powerOf2 sized arrays
        if (nIsPow2 || i + blockSize < n)
            mySum += g_idata[i+blockSize];

        i += gridSize;
    }

    // each thread puts its local sum into shared memory
    sdata[tid] = mySum;
    cg::sync(cta);


    // do reduction in shared mem
    if ((blockSize >= 512) && (tid < 256))
    {
        sdata[tid] = mySum = mySum + sdata[tid + 256];
    }

    cg::sync(cta);

    if ((blockSize >= 256) &&(tid < 128))
    {
            sdata[tid] = mySum = mySum + sdata[tid + 128];
    }

    cg::sync(cta);

    if ((blockSize >= 128) && (tid <  64))
    {
       sdata[tid] = mySum = mySum + sdata[tid +  64];
    }

    cg::sync(cta);

    cg::thread_block_tile<32> tile32 = cg::tiled_partition<32>(cta);

    if (cta.thread_rank() < 32)
    {
        // Fetch final intermediate sum from 2nd warp
        if (blockSize >=  64) mySum += sdata[tid + 32];
        // Reduce final warp using shuffle
        for (int offset = tile32.size()/2; offset > 0; offset /= 2) 
        {
             mySum += tile32.shfl_down(mySum, offset);
        }
    }

    // write result for this block to global mem
    if (cta.thread_rank() == 0) g_odata[blockIdx.x] = mySum;
}


extern "C"
bool isPow2(unsigned int x);

__global__
void sq(int n, float *x, float *y){
  /* Write me -- kernel */
  for (size_t i = 0; i < n; ++i)
  {
    y[i] = x[i] * x[i];
  }
}

int main(int argc, char* argv[]) {

  size_t exponent           = 27;
  size_t num_trips          = 1;

  if (argc >= 2) exponent   = std::stol(argv[1]);
  if (argc >= 3) num_trips  = std::stol(argv[2]);

  size_t N = 1 << exponent;
  int const block_size = 256;
  int num_blocks = (N + block_size - 1) / block_size;

  float *x = nullptr, *y = nullptr, *result = nullptr;

  hipMallocManaged(&x, N*sizeof(float));
  hipMallocManaged(&y, num_blocks*sizeof(float));
  hipMallocManaged(&result, 1*sizeof(float));

  for (int i = 0; i < N; i++) {
    x[i] = i;
  }

  DEF_TIMER(cuda_norm);
  START_TIMER(cuda_norm);
  hipDeviceSynchronize();
  for (size_t i = 0; i < num_trips; ++i) {
    sq<<<num_blocks, block_size>>>(N, x, y);
    //reduce6<float, block_size, true><<<num_blocks, block_size>>>(y, result, N);
    reduce0<float><<<num_blocks, block_size>>>(y, result, N);
    hipDeviceSynchronize();
  }


  double cuda_time = STOP_TIMER_QUIETLY(cuda_norm);
  std::cout << exponent << "\t" << num_trips << "\t" << cuda_time << "\t" << *result << std::endl;

  hipFree(x); hipFree(y);
  
  return 0;
}
